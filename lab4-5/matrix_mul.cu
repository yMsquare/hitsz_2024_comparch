// #define USE_CUBLAS

// #include <__clang_cuda_builtin_vars.h>
#include <cstdio>
#include <hip/hip_runtime.h>
#include <iostream>
#ifdef USE_CUBLAS
#include <cublas_v2.h>
#endif
#include <cmath>

using namespace std;

const int TILE_WIDTH = 16; // 定义块block大小

// /////////
// // Matrix multiplication with shared memory (CUDA Kernel) on the device: C =
// A * B
// /////////
const int BLOCK_SIZE = TILE_WIDTH;
__global__ void MatrixMulSharedMemKernel(float *A, float *B, float *C, int wA,
                                         int wB) {
  // Block index
  int bx = blockIdx.x;
  int by = blockIdx.y;

  // Thread index
  int tx = threadIdx.x;
  int ty = threadIdx.y;

  // Index of the first sub-matrix of A processed by the block
  int aBegin = wA * BLOCK_SIZE * by;

  // Index of the last sub-matrix of A processed by the block
  int aEnd = aBegin + wA - 1;

  // Step size used to iterate through the sub-matrices of A
  int aStep = BLOCK_SIZE;

  // Index of the first sub-matrix of B processed by the block
  int bBegin = BLOCK_SIZE * bx;

  // Step size used to iterate through the sub-matrices of B
  int bStep = BLOCK_SIZE * wB;

  // Csub is used to store the element of the block sub-matrix
  // that is computed by the thread
  float Csub = 0;

  // Loop over all the sub-matrices of A and B
  // required to compute the block sub-matrix
  for (int a = aBegin, b = bBegin; a < aEnd; a += aStep, b += bStep) {
    // Declaration of the shared memory array As used to
    // store the sub-matrix of A
    __shared__ float As[BLOCK_SIZE][BLOCK_SIZE];

    // Declaration of the shared memory array Bs used to
    // store the sub-matrix of B
    __shared__ float Bs[BLOCK_SIZE][BLOCK_SIZE];

    // Load the matrices from device memory
    // to shared memory; each **thread** loads
    // one element of each matrix
    // --- TO DO :Load the elements of the sub-matrix of A into As ---
    // ---        Load the elements of the sub-matrix of B into Bs ---
    //  if (a + ty * wA + tx < aEnd) {
    //     As[ty][tx] = A[a + ty * wA + tx];
    //   } else {
    //     As[ty][tx] = 0.0f;
    //   }
    int aIndex = a + ty * wA + tx;
    if ((aIndex / wA) < wA && (aIndex % wA) < wA) {
      As[ty][tx] = A[aIndex];
    } else {
      As[ty][tx] = 0.0f;
    }

    int bIndex = b + ty * wB + tx;
    if ((bIndex / wB) < wB && (bIndex % wB) < wB) {
      Bs[ty][tx] = B[bIndex];
    } else {
      Bs[ty][tx] = 0.0f;
    }

    // NOTE: Ensure that the thread indices do not exceed the matrix dimensions
    // to avoid out-of-bounds access.
    //       Use boundary checks to load valid elements into shared memory, and
    //       set invalid elements to 0.0f

    // Synchronize to make sure the matrices are loaded
    __syncthreads();

    // Multiply the two matrices together;
    // each thread computes one element
    // of the block sub-matrix
#pragma unroll
    // --- TO DO :Implement the matrix multiplication using the sub-matrices As
    // and Bs ---
    for (int k = 0; k < BLOCK_SIZE; k++) {
      Csub += As[ty][k] * Bs[k][tx];
    }
    // Synchronize to make sure that the preceding
    // computation is done before loading two new
    // sub-matrices of A and B in the next iteration
    __syncthreads();
  }

  // Write the block sub-matrix to device memory;
  // each thread writes one element
  int c = wB * BLOCK_SIZE * by + BLOCK_SIZE * bx;
  if ((ty + BLOCK_SIZE * by) < wA && (tx + BLOCK_SIZE * bx) < wB) {
    C[c + ty * wB + tx] = Csub;
  }
  // --- TO DO :Store the computed Csub result into matrix C ---
  // NOTE: Ensure that the thread indices "c" do not exceed the matrix
  // dimensions to avoid out-of-bounds access.
  //       Use boundary checks to write valid elements to the output matrix C.
}

//! For square matrices only
__global__ void MatrixMulKernel(float *d_M, float *d_N, float *d_P, int width) {
  // Calculate the row index of the P element and M
  // *** TO DO: Compute the row index for the current thread ***
  int row = blockIdx.y * blockDim.y + threadIdx.y;

  // Calculate the column index of the P element and N
  // *** TO DO: Compute the column index for the current thread ***
  int col = blockIdx.x * blockDim.x + threadIdx.x;

  // Ensure the thread is within bounds
  if ((row < width) && (col < width)) {
    float pValue = 0.0;
    for (int k = 0; k < width; k++) {
      pValue += d_M[row * width + k] * d_N[col * width + col];
    }
    d_P[row * width + col] = pValue;

    // Each thread computes one element of the matrix
    // *** TO DO: Implement the matrix multiplication for a single element ***

    // Store the computed value into the output matrix
    // *** TO DO: Write the computed value to the correct position in d_P ***
    // d_P[row * width + col] = ...;
  }
}

////////////////////////////////////////////////////////////////////////////////
//! Compute reference data set matrix multiply on CPU
//! C = A * B
//! @param C          reference data, computed but preallocated
//! @param A          matrix A as provided to device
//! @param B          matrix B as provided to device
//! @param hA         height of matrix A
//! @param wA         width of matrix A
//! @param wB         width of matrix B
////////////////////////////////////////////////////////////////////////////////
void matrixMulCPU(float *C, const float *A, const float *B, unsigned int hA,
                  unsigned int wA, unsigned int wB) {
  for (unsigned int i = 0; i < hA; ++i)
    for (unsigned int j = 0; j < wB; ++j) {
      double sum = 0;

      for (unsigned int k = 0; k < wA; ++k) {
        double a = A[i * wA + k];
        double b = B[k * wB + j];
        sum += a * b;
      }

      C[i * wB + j] = (float)sum;
    }
}

void printDiff(float *data1, float *data2, int width, int height,
               int iListLength, float fListTol) {
  printf("Listing first %d Differences > %.6f...\n", iListLength, fListTol);
  int i, j, k;
  int error_count = 0;

  for (j = 0; j < height; j++) {
    for (i = 0; i < width; i++) {
      k = j * width + i;
      float fDiff = fabs(data1[k] - data2[k]);

      if (fDiff > fListTol) {
        if (error_count < iListLength) {
          printf("    Loc(%d,%d)\tCPU=%.5f\tGPU=%.5f\tDiff=%.6f\n", i, j,
                 data1[k], data2[k], fDiff);
        }

        error_count++;
      }
    }
  }

  printf(" \n  Total Errors = %d\n", error_count);
}

void getArg(int argc, char *argv[], int &size, int &check) {
  if (argc != 3) {
    cerr << "Usage: " << argv[0] << " <check_enable> <size>\n";
    cerr << "\tcheck_enable: 1 to enable result checking\n";
    cerr << "\tsize: size of the matrix\n";
    exit(1);
  }

  int val1, val2;
  try {
    val1 = stoi(argv[1]);
    val2 = stoi(argv[2]);
  } catch (const invalid_argument &e) {
    cerr << "ERROR: parameters should be integer\n";
    exit(1);
  }

  check = val1;
  size = val2;
}

int main(int argc, char *argv[]) {
  int size, check;
  getArg(argc, argv, size, check);

  int m = size, n = size, k = size;

  // 声明存放在GPU上的数组
  float *h_M, *h_N, *d_M, *d_N;
  float *h_P, *d_P;

  size_t sizeM = m * k * sizeof(float);
  size_t sizeN = k * n * sizeof(float);
  size_t sizeP = m * n * sizeof(float);

  // Allocate host memory
  h_M = (float *)malloc(sizeM);
  h_N = (float *)malloc(sizeN);
  h_P = (float *)malloc(sizeP);
  float *reference = (float *)malloc(sizeP);

  // Allocate device memory
  hipMalloc(&d_M, sizeM);
  hipMalloc(&d_N, sizeN);
  hipMalloc(&d_P, sizeP);

  // Init data
  for (int i = 0; i < m * n; ++i) {
    if (i % 2 == 0)
      h_M[i] = 1.0;
    else
      h_M[i] = 0.5;
  }

  for (int i = 0; i < n * k; ++i) {
    if (i % 2 == 0)
      h_N[i] = 0.5;
    else
      h_N[i] = 1.0;
  }

  // Copy data from CPU to GPU
  hipMemcpy(d_M, h_M, sizeM, hipMemcpyHostToDevice);
  hipMemcpy(d_N, h_N, sizeN, hipMemcpyHostToDevice);

  // Timing records
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start, 0);

  // Launch kernel 定义grid&block
  dim3 grid((int)ceil(k * 1.0 / TILE_WIDTH), (int)ceil(m * 1.0 / TILE_WIDTH));
  dim3 block(TILE_WIDTH, TILE_WIDTH);

  int nIter = 5;
#ifdef USE_CUBLAS
  cublasHandle_t handle;
  cublasCreate(&handle);
#endif
  const float alpha = 1.0f;
  const float beta = 0.0f;
  for (int j = 0; j < nIter; j++) {
    // matrixMulCPU(reference, h_M, h_N, m, k, n);
    //  MatrixMulKernel<<<grid, block>>>(d_M, d_N, d_P, m);
    MatrixMulSharedMemKernel<<<grid, block>>>(d_M, d_N, d_P, m, n);
    // cublasSgemm(handle, CUBLAS_OP_N, CUBLAS_OP_N, n, m, k, &alpha, d_N, n,
    // d_M, k, &beta, d_P, n);
  }

  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  float msecPerMatrixMul;
  hipEventElapsedTime(&msecPerMatrixMul, start, stop);
  msecPerMatrixMul /= nIter;
  printf("Kernel Elpased Time: %.3f ms\n", msecPerMatrixMul);

  // Compute and print the performance
  double flopsPerMatrixMul = 2.0 * (double)m * (double)n * (double)k;
  double gigaFlops =
      (flopsPerMatrixMul * 1.0e-9f) / (msecPerMatrixMul / 1000.0f);
  printf("Performance= %.2f GFlop/s, Time= %.3f msec, Size= %.0f Ops\n",
         gigaFlops, msecPerMatrixMul, flopsPerMatrixMul);

  // Copy data from GPU to CPU
  hipMemcpy(h_P, d_P, sizeP, hipMemcpyDeviceToHost);

  // compute reference solution
  if (check == 1) {
    printf("Computing result using host CPU...");
    matrixMulCPU(reference, h_M, h_N, m, k, n);
    printf("done.\n");
    printDiff(reference, h_P, n, m, 100, 1.0e-5f);
  }

  free(h_P);
  free(h_M);
  free(h_N);
  hipFree(d_P);
  hipFree(d_M);
  hipFree(d_N);
#ifdef USE_CUBLAS
  cublasDestroy(handle);
#endif

  return 0;
}
